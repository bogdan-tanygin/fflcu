#include "hip/hip_runtime.h"
/*
  Copyright (C) 2012 Alexander (Polyakov) Peletskyi

  This file is part of FFLCU.

  FFLCU is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  FFLCU is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/*
 * fflComputeBH.cu
 *
 *  Created on: Jan 31, 2012
 *      Author: alexander
 */

#include "fflComputeBH.cuh"

#define SQ(x) ((x)*(x))
#define IND (blockDim.x * blockIdx.x + threadIdx.x)
#define DEBUG
const float myu0 = 4e-7 * M_PI;
const float g = 9.81;

/*__device__ template <typename T> int sgn(T val)
{
    return (val > T(0)) - (val < T(0));
}
*/
__device__ float sgn(float x){
	return (x > 0.0f) ? 1.0f : ((x < 0.0f) ? -1.0f : 0.0f);
}


using namespace std;

__constant__ int nnodesd, nbodiesd;
__constant__ volatile float epssqd, itolsqd;

__device__ volatile int bottomd, maxdepthd, blkcntd;
__device__ volatile float radiusd;

__constant__ ConstParams devCParBH;
__constant__ ChangableParams devChParBH;

__constant__ volatile float* xd;
__constant__ volatile float* yd;
__constant__ volatile float* zd;
__constant__ volatile float* phyd;
__constant__ volatile float* thetad;
__constant__ volatile float* uxd;
__constant__ volatile float* uyd;
__constant__ volatile float* uzd;

__constant__ volatile float* phy1d; //=d(phy)/(dt)
__constant__ volatile float* theta1d; //=d(theta)/(dt)

__constant__ volatile float* deltaPhyd;
__constant__ volatile float* deltaThetad;

__constant__ volatile float* x1d;
__constant__ volatile float* y1d;
__constant__ volatile float* z1d;

__constant__ volatile float* deltaXd;
__constant__ volatile float* deltaYd;
__constant__ volatile float* deltaZd;

__constant__ volatile float* massd;

__constant__ volatile int *errd;
__constant__ volatile int *sortd;
__constant__ volatile int *childd;
__constant__ volatile int *countd;
__constant__ volatile int *startd;

__constant__ volatile float *minxd;
__constant__ volatile float *maxxd;
__constant__ volatile float *minyd;
__constant__ volatile float *maxyd;
__constant__ volatile float *minzd;
__constant__ volatile float *maxzd;

__constant__ hiprandState *rndStatesd;

__global__ __launch_bounds__(THREADS0, FACTOR0)
void averageUKernel(float3* u)
{
	register int i, inc, cntr;
	float sinTheta, sinPhy, cosTheta, cosPhy;
	__shared__ float ux[THREADS0], uy[THREADS0], uz[THREADS0];
	__shared__ int shCntr;

	if (threadIdx.x + blockIdx.x * blockDim.x == 0) {
		u->x = 0.0f;
		u->y = 0.0f;
		u->z = 0.0f;
	}

	if (threadIdx.x == 0) {
		shCntr = 0;
	}

	ux[threadIdx.x] = 0.0f;
	uy[threadIdx.x] = 0.0f;
	uz[threadIdx.x] = 0.0f;

	// iterate over all nodes assigned to thread
	inc = blockDim.x * gridDim.x;
	cntr = 0;
	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
		sincos(thetad[i], &sinTheta, &cosTheta);
		sincos(phyd[i], &sinPhy, &cosPhy);
		ux[threadIdx.x] += sinTheta * cosPhy;
		uy[threadIdx.x] += sinTheta * sinPhy;
		uz[threadIdx.x] += cosTheta;
		cntr++;
	}

	atomicAdd(&shCntr, cntr);

	if (cntr > 0) {
		ux[threadIdx.x] = ux[threadIdx.x] / ((float)cntr);
		uy[threadIdx.x] = uy[threadIdx.x] / ((float)cntr);
		uz[threadIdx.x] = uz[threadIdx.x] / ((float)cntr);
	}

	__syncthreads();
	//cntr = 0;
	for (cntr = THREADS0 / 2; cntr > 0; cntr /= 2 ) {
		if (threadIdx.x < cntr) {
			if(threadIdx.x + blockIdx.x * blockDim.x + cntr < nbodiesd){
				ux[threadIdx.x] = 0.5 * (ux[threadIdx.x] + ux[threadIdx.x + cntr]);
				uy[threadIdx.x] = 0.5 * (uy[threadIdx.x] + uy[threadIdx.x + cntr]);
				uz[threadIdx.x] = 0.5 * (uz[threadIdx.x] + uz[threadIdx.x + cntr]);
			}
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		if (shCntr > 0){
			ux[0] *= (float)shCntr / (float)nbodiesd;
			uy[0] *= (float)shCntr / (float)nbodiesd;
			uz[0] *= (float)shCntr / (float)nbodiesd;
			atomicAdd(&(u->x), ux[0]);
			atomicAdd(&(u->y), uy[0]);
			atomicAdd(&(u->z), uz[0]);
		}
	}
}

/******************************************************************************/
/*** initialize memory ********************************************************/
/******************************************************************************/

__global__ void initializationKernel(hiprandState *state, unsigned long seed)
{
	register int ind;
	ind = IND;
	if (ind == 0) {
		*errd = 0;
		maxdepthd = 1;
		blkcntd = 0;
	}

	hiprand_init (seed, ind, 0, &state[ind]);
}



__global__
__launch_bounds__(THREADS0, FACTOR0)
void calcUKernel()
{
	register int i, inc;
	float sinTheta, sinPhy, cosTheta, cosPhy;
	// iterate over all nodes assigned to thread
	inc = blockDim.x * gridDim.x;
	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
		sincos(thetad[i], &sinTheta, &cosTheta);
		sincos(phyd[i], &sinPhy, &cosPhy);
		uxd[i] = sinTheta * cosPhy;
		uyd[i] = sinTheta * sinPhy;
		uzd[i] = cosTheta;
	}
}

/******************************************************************************/
/*** compute center and radius ************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS1, FACTOR1)
void boundingBoxKernel()
{
	register int i, j, k, inc;
	register float val, minx, maxx, miny, maxy, minz, maxz;
	__shared__ volatile float sminx[THREADS1], smaxx[THREADS1], sminy[THREADS1], smaxy[THREADS1], sminz[THREADS1], smaxz[THREADS1];

	// initialize with valid data (in case #bodies < #threads)
	minx = maxx = xd[0];
	miny = maxy = yd[0];
	minz = maxz = zd[0];

	// scan all bodies
	i = threadIdx.x;
	inc = THREADS1 * gridDim.x;
	for (j = i + blockIdx.x * THREADS1; j < nbodiesd; j += inc) {
		val = xd[j];
		minx = min(minx, val);
		maxx = max(maxx, val);
		val = yd[j];
		miny = min(miny, val);
		maxy = max(maxy, val);
		val = zd[j];
		minz = min(minz, val);
		maxz = max(maxz, val);
	}

	// reduction in shared memory
	sminx[i] = minx;
	smaxx[i] = maxx;
	sminy[i] = miny;
	smaxy[i] = maxy;
	sminz[i] = minz;
	smaxz[i] = maxz;

	for (j = THREADS1 / 2; j > 0; j /= 2) {
		__syncthreads();
		if (i < j) {
			k = i + j;
			sminx[i] = minx = min(minx, sminx[k]);
			smaxx[i] = maxx = max(maxx, smaxx[k]);
			sminy[i] = miny = min(miny, sminy[k]);
			smaxy[i] = maxy = max(maxy, smaxy[k]);
			sminz[i] = minz = min(minz, sminz[k]);
			smaxz[i] = maxz = max(maxz, smaxz[k]);
		}
	}

	// write block result to global memory
	if (i == 0) {
		k = blockIdx.x;
		minxd[k] = minx;
		maxxd[k] = maxx;
		minyd[k] = miny;
		maxyd[k] = maxy;
		minzd[k] = minz;
		maxzd[k] = maxz;

		inc = gridDim.x - 1;
		if (inc == atomicInc((unsigned int *)&blkcntd, inc)) {
		// I'm the last block, so combine all block results
			for (j = 0; j <= inc; j++) {
				minx = min(minx, minxd[j]);
				maxx = max(maxx, maxxd[j]);
				miny = min(miny, minyd[j]);
				maxy = max(maxy, maxyd[j]);
				minz = min(minz, minzd[j]);
				maxz = max(maxz, maxzd[j]);
			}

			// compute 'radius'
			val = max(maxx - minx, maxy - miny);
			radiusd = max(val, maxz - minz) * 0.5f;

			// create root node
			k = nnodesd;
			bottomd = k;

			massd[k] = -1.0f;
			startd[k] = 0;
			xd[k] = (minx + maxx) * 0.5f;
			yd[k] = (miny + maxy) * 0.5f;
			zd[k] = (minz + maxz) * 0.5f;
			k *= 8;
			for (i = 0; i < 8; i++) childd[k + i] = -1;
		}
	}
}


/******************************************************************************/
/*** build tree ***************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS2, FACTOR2)
void treeBuildingKernel()
{
	register int i, j, k, depth, localmaxdepth, skip, inc;
	register float x, y, z, r;
	register float px, py, pz;
	register int ch, n, cell, locked, patch;
	register float radius, rootx, rooty, rootz;

	// cache root data
	radius = radiusd;
	rootx = xd[nnodesd];
	rooty = yd[nnodesd];
	rootz = zd[nnodesd];

	localmaxdepth = 1;
	skip = 1;
	inc = blockDim.x * gridDim.x;
	i = threadIdx.x + blockIdx.x * blockDim.x;

	// iterate over all bodies assigned to thread
	while (i < nbodiesd) {
		if (skip != 0) {
			// new body, so start traversing at root
			skip = 0;
			px = xd[i];
			py = yd[i];
			pz = zd[i];
			n = nnodesd;
			depth = 1;
			r = radius;
			j = 0;
			// determine which child to follow
			if (rootx < px) j = 1;
			if (rooty < py) j += 2;
			if (rootz < pz) j += 4;
		}

		// follow path to leaf cell
		ch = childd[n * 8 + j];
		while (ch >= nbodiesd) {
			n = ch;
			depth++;
			r *= 0.5f;
			j = 0;
			// determine which child to follow
			if (xd[n] < px) j = 1;
			if (yd[n] < py) j += 2;
			if (zd[n] < pz) j += 4;
			ch = childd[n * 8 + j];
		}

		if (ch != -2) {	// skip if child pointer is locked and try again later
			locked = n  * 8 + j;
			if (ch == atomicCAS((int *)&childd[locked], ch, -2)) {	// try to lock
				if (ch == -2) {
					printf("!!!!!!!Error: ch = -2\n");
					break;
				}
				if (ch == -1) {
					// if null, just insert the new body
					childd[locked] = i;
				} else {	// there already is a body in this position
					patch = -1;
					// create new cell(s) and insert the old and new body
					do {
						depth++;

						cell = atomicSub((int *)&bottomd, 1) - 1;
						if (cell <= nbodiesd) {
							*errd = 1;
							bottomd = nnodesd;
						}
						patch = max(patch, cell);

						x = (j & 1) * r;
						y = ((j >> 1) & 1) * r;
						z = ((j >> 2) & 1) * r;
						r *= 0.5f;

						massd[cell] = -1.0f;
						startd[cell] = -1;
						x = xd[cell] = xd[n] - r + x;
						y = yd[cell] = yd[n] - r + y;
						z = zd[cell] = zd[n] - r + z;
						for (k = 0; k < 8; k++)
							childd[cell* 8 + k] = -1;

						if (patch != cell) {
							childd[n * 8 + j] = cell;
						}

						j = 0;
						if (x < xd[ch]) j = 1;
						if (y < yd[ch]) j += 2;
						if (z < zd[ch]) j += 4;
						childd[cell * 8 + j] = ch;

						n = cell;
						j = 0;
						if (x < px) j = 1;
						if (y < py) j += 2;
						if (z < pz) j += 4;

						ch = childd[n * 8 + j];
						// repeat until the two bodies are different children
					} while (ch >= 0);

					childd[n * 8 + j] = i;
					__threadfence();	// push out subtree
					childd[locked] = patch;
				}

				localmaxdepth = max(depth, localmaxdepth);
				i += inc;	// move on to next body
				skip = 1;
			}
		}
		__syncthreads();	// throttle
	}
	// record maximum tree depth
	atomicMax((int *)&maxdepthd, localmaxdepth);
#ifdef DEBUG
//	if ( maxdepthd == localmaxdepth ) printf ("Max tree depth: %d\n", maxdepthd);
#endif
}


__global__ void __treeBuildingKernel()
{
  register int i, j, k, depth, localmaxdepth, skip, inc;
  register float x, y, z, r;
  register float px, py, pz;
  register int ch, n, cell, locked, patch;
  __shared__ float radius, rootx, rooty, rootz;

  i = threadIdx.x;
  if (i == 0) {
    // cache root data
    radius = radiusd;
    rootx = xd[nnodesd];
    rooty = yd[nnodesd];
    rootz = zd[nnodesd];
  }
  __syncthreads();

  localmaxdepth = 1;
  skip = 1;
  inc = blockDim.x * gridDim.x;
  i += blockIdx.x * blockDim.x;

  // iterate over all bodies assigned to thread
  while (i < nbodiesd) {
    if (skip != 0) {
      // new body, so start traversing at root
      skip = 0;
      px = xd[i];
      py = yd[i];
      pz = zd[i];
      n = nnodesd;
      depth = 1;
      r = radius;
      j = 0;
      // determine which child to follow
      if (rootx < px) j = 1;
      if (rooty < py) j += 2;
      if (rootz < pz) j += 4;
    }

    ch = childd[n*8+j];
    // follow path to leaf cell
    while (ch >= nbodiesd) {
      n = ch;
      depth++;
      r *= 0.5f;
      j = 0;
      // determine which child to follow
      if (xd[n] < px) j = 1;
      if (yd[n] < py) j += 2;
      if (zd[n] < pz) j += 4;
      ch = childd[n*8+j];
    }

    if (ch != -2) {  // skip if child pointer is locked and try again later
      locked = n*8+j;
      if (ch == atomicCAS((int*)&childd[locked], ch, -2)) {  // try to lock
        if (ch == -1) {
          // if null, just insert the new body
          childd[locked] = i;
        } else {  // there already is a body in this position
          patch = -1;
          // create new cell(s) and insert the old and new body
          do {
            depth++;

            cell = atomicSub((int*)&bottomd, 1) - 1;
            if (cell <= nbodiesd) {
              *errd = 1;
              bottomd = nnodesd;
            }
            patch = max(patch, cell);

            x = (j & 1) * r;
            y = ((j >> 1) & 1) * r;
            z = ((j >> 2) & 1) * r;
            r *= 0.5f;

            massd[cell] = -1.0f;
            startd[cell] = -1;
            x = xd[cell] = xd[n] - r + x;
            y = yd[cell] = yd[n] - r + y;
            z = zd[cell] = zd[n] - r + z;
#pragma unroll 8
            for (k = 0; k < 8; k++) childd[cell*8+k] = -1;

            if (patch != cell) {
              childd[n*8+j] = cell;
            }

            j = 0;
            if (x < xd[ch]) j = 1;
            if (y < yd[ch]) j += 2;
            if (z < zd[ch]) j += 4;
            childd[cell*8+j] = ch;

            n = cell;
            j = 0;
            if (x < px) j = 1;
            if (y < py) j += 2;
            if (z < pz) j += 4;

            ch = childd[n*8+j];
            // repeat until the two bodies are different children
          } while (ch >= 0);
          childd[n*8+j] = i;
          __threadfence();
          childd[locked] = patch;
        }

        localmaxdepth = max(depth, localmaxdepth);
        i += inc;  // move on to next body
        skip = 1;
      }
    }
    __syncthreads();
  }
  atomicMax((int*)&maxdepthd, localmaxdepth);
}


/******************************************************************************/
/*** compute center of mass ***************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS3, FACTOR3)
void summarizationKernel()
{
	register int i, j, k, ch, inc, missing, cnt, bottom;
	register float m, cm, px, py, pz, ux, uy, uz;
	__shared__ volatile int child[THREADS3 * 8];

	bottom = bottomd;
	inc = blockDim.x * gridDim.x;
	k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;	// align to warp size
	if (k < bottom) k += inc;

	missing = 0;
	//int iteration = 0;
	// iterate over all cells assigned to thread
	while (k <= nnodesd) {
		//iteration++;
		if (missing == 0) {
			// new cell, so initialize
			cm = 0.0f;
			px = 0.0f;
			py = 0.0f;
			pz = 0.0f;
			ux = 0.0f;
			uy = 0.0f;
			uz = 0.0f;
			cnt = 0;
			j = 0;
			for (i = 0; i < 8; i++) {
				ch = childd[k * 8 + i];
				if (ch >= 0) {
			if (i != j) {
				// move children to front (needed later for speed)
				childd[k* 8 + i] = -1;
				childd[k* 8 + j] = ch;
			}
			child[missing * THREADS3 + threadIdx.x] = ch;	// cache missing children
			m = massd[ch];
			missing++;
			if (m >= 0.0f) {
				// child is ready
				missing--;
				if (ch >= nbodiesd) {	// count bodies (needed later)
					cnt += countd[ch] - 1;
				}
				// add child's contribution
				cm += m;
				px += xd[ch] * m;
				py += yd[ch] * m;
				pz += zd[ch] * m;
				ux += uxd[ch];
				uy += uyd[ch];
				uz += uzd[ch];
			}
			j++;
				}
			}
			cnt += j;
		}

		if (missing != 0) {
			do {
				// poll missing child
				ch = child[(missing - 1) * THREADS3 + threadIdx.x];
				m = massd[ch];
				if (m >= 0.0f) {
					// child is now ready
					missing--;
					if (ch >= nbodiesd) {
						// count bodies (needed later)
						cnt += countd[ch] - 1;
					}
					// add child's contribution
					cm += m;
					px += xd[ch] * m;
					py += yd[ch] * m;
					pz += zd[ch] * m;
					ux += uxd[ch];
					uy += uyd[ch];
					uz += uzd[ch];
				}
				// repeat until we are done or child is not ready
			} while ((m >= 0.0f) && (missing != 0));
		}

		if (missing == 0) {
			// all children are ready, so store computed information
	//		if (m > 1E-5){
				countd[k] = cnt;
				m = 1.0f / cm;
				xd[k] = px * m;
				yd[k] = py * m;
				zd[k] = pz * m;

				uxd[k] = ux;
				uyd[k] = uy;
				uzd[k] = uz;
	/*		} else {
				printf ("Warning - zero cell in summation");
				countd[k] = cnt;
				m = 0.0f;
				xd[k] = px * m;
				yd[k] = py * m;
				zd[k] = pz * m;

				uxd[k] = 0.0f;
				uyd[k] = 0.0f;
				uzd[k] = 0.0f;
			}*/
			__threadfence();	// make sure data are visible before setting mass
			massd[k] = cm;
			k += inc;	// move on to next cell
		}
	/*	if (iteration > 100000){
			printf ("To much iterations");
			printf ("k = %d, nnodes = %d", k, nnodesd);
			break;
		}*/
	}	//while
}


__global__ void __summarizationKernel()
{
  register int i, j, k, ch, inc, missing, cnt;
  register float m, cm, px, py, pz, ux, uy, uz;
  __shared__ int bottom, child[THREADS3 * 8];

  if (0 == threadIdx.x) {
    bottom = bottomd;
  }
  __syncthreads();

  inc = blockDim.x * gridDim.x;
  k = (bottom & (-WARPSIZE)) + threadIdx.x + blockIdx.x * blockDim.x;  // align to warp size
  if (k < bottom) k += inc;

  missing = 0;
  // iterate over all cells assigned to thread
  while (k <= nnodesd) {
    if (missing == 0) {
      // new cell, so initialize
		cm = 0.0f;
		px = 0.0f;
		py = 0.0f;
		pz = 0.0f;
		ux = 0.0f;
		uy = 0.0f;
		uz = 0.0f;
		cnt = 0;
		j = 0;
#pragma unroll 8
      for (i = 0; i < 8; i++) {
        ch = childd[k*8+i];
        if (ch >= 0) {
          if (i != j) {
            // move children to front (needed later for speed)
            childd[k*8+i] = -1;
            childd[k*8+j] = ch;
          }
          child[missing*THREADS3+threadIdx.x] = ch;  // cache missing children
          m = massd[ch];
          missing++;
          if (m >= 0.0f) {
            // child is ready
            missing--;
            if (ch >= nbodiesd) {  // count bodies (needed later)
              cnt += countd[ch] - 1;
            }
            // add child's contribution
			cm += m;
			px += xd[ch] * m;
			py += yd[ch] * m;
			pz += zd[ch] * m;
			ux += uxd[ch];
			uy += uyd[ch];
			uz += uzd[ch];;
          }
          j++;
        }
      }
      cnt += j;
    }

    if (missing != 0) {
      do {
        // poll missing child
        ch = child[(missing-1)*THREADS3+threadIdx.x];
        m = massd[ch];
        if (m >= 0.0f) {
          // child is now ready
          missing--;
          if (ch >= nbodiesd) {
            // count bodies (needed later)
            cnt += countd[ch] - 1;
          }
          // add child's contribution
			cm += m;
			px += xd[ch] * m;
			py += yd[ch] * m;
			pz += zd[ch] * m;
			ux += uxd[ch];
			uy += uyd[ch];
			uz += uzd[ch];
        }
        // repeat until we are done or child is not ready
      } while ((m >= 0.0f) && (missing != 0));
    }

    if (missing == 0) {
      // all children are ready, so store computed information
		countd[k] = cnt;
		m = 1.0f / cm;
		xd[k] = px * m;
		yd[k] = py * m;
		zd[k] = pz * m;

		uxd[k] = ux;
		uyd[k] = uy;
		uzd[k] = uz;
      __threadfence();
      massd[k] = cm;
      k += inc;  // move on to next cell
    }
  }
}


/******************************************************************************/
/*** sort bodies **************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS4, FACTOR4)
void sortKernel()
{
	register int i, k, ch, dec, start, bottom;

	bottom = bottomd;
	dec = blockDim.x * gridDim.x;
	k = nnodesd + 1 - dec + threadIdx.x + blockIdx.x * blockDim.x;

	// iterate over all cells assigned to thread
	while (k >= bottom) {
		start = startd[k];
		if (start >= 0) {
			for (i = 0; i < 8; i++) {
				ch = childd[k* 8 + i];
				if (ch >= nbodiesd) {
			// child is a cell
			startd[ch] = start;	// set start ID of child
			start += countd[ch];	// add #bodies in subtree
				} else if (ch >= 0) {
			// child is a body
			sortd[start] = ch;	// record body in 'sorted' array
			start++;
				}
			}
			k -= dec;	// move on to next cell
		}
		__syncthreads();	// throttle
	}
}


/******************************************************************************/
/*** compute force ************************************************************/
/******************************************************************************/

__global__
__launch_bounds__(THREADS5, FACTOR5)
void forceCalculationKernel()
{
	register int i, j, k, n, depth, base, sbase, diff, t;
	register float px, py, pz, dx, dy, dz, tmp, fx, fy, fz, hx, hy, hz, ux, uy, uz;
	register float ucx, ucy, ucz;
	__shared__ volatile int pos[MAXDEPTH * THREADS5/WARPSIZE], node[MAXDEPTH * THREADS5/WARPSIZE];
	__shared__ float dq[MAXDEPTH * THREADS5/WARPSIZE];
	hiprandState localState;
	//localState = rndStatesd[IND];

	float b, b2, d1, dd5;
	float bb2d7, umd5;
	float pow3s2d2, flj;

	if (0 == threadIdx.x) {
		tmp = radiusd;
		// precompute values that depend only on tree level
		dq[0] = tmp * tmp * itolsqd;
		for (i = 1; i < maxdepthd; i++) {
			dq[i] = dq[i - 1] * 0.25f;
			dq[i - 1] += epssqd;
		}
		dq[i - 1] += epssqd;

		if (maxdepthd > MAXDEPTH) {
			*errd = maxdepthd;
		}
	}
	__syncthreads();

	if (maxdepthd <= MAXDEPTH) {
		// figure out first thread in each warp (lane 0)
		base = threadIdx.x / WARPSIZE;
		sbase = base * WARPSIZE;
		j = base * MAXDEPTH;

		diff = threadIdx.x - sbase;
		// make multiple copies to avoid index calculations later
		if (diff < MAXDEPTH) {
			dq[diff+j] = dq[diff];
		}
		__syncthreads();

		// iterate over all bodies assigned to thread
		for (k = threadIdx.x + blockIdx.x * blockDim.x; k < nbodiesd; k += blockDim.x * gridDim.x) {
			i = sortd[k];	// get permuted/sorted index
			// cache position info
			px = xd[i];
			py = yd[i];
			pz = zd[i];

			ux = uxd[i];
			uy = uyd[i];
			uz = uzd[i];

			hx = 0.0f;
			hy = 0.0f;
			hz = 0.0f;

			fx = 0.0f;
			fy = 0.0f;
			fz = 0.0f;

			// initialize iteration stack, i.e., push root node onto stack
			depth = j;
			if (sbase == threadIdx.x) {
				node[j] = nnodesd;
				pos[j] = 0;
			}

			while (depth >= j) {
				// stack is not empty
				while ((t = pos[depth]) < 8) {
					// node on top of stack has more children to process
					n = childd[node[depth] * 8 + t];	// load child pointer
					if (sbase == threadIdx.x) {
						// I'm the first thread in the warp
						pos[depth] = t + 1;
					}
					if (n >= 0) {
						dx = -(xd[n] - px);
						dy = -(yd[n] - py);
						dz = -(zd[n] - pz);
						tmp = dx * dx + (dy * dy + dz * dz);	// compute distance squared
						if ((n < nbodiesd) || __all(tmp >= dq[depth])) {	// check if all threads agree that cell is far enough away (or is a body)
							if (n != i) {

								ucx = uxd[n];
								ucy = uyd[n];
								ucz = uzd[n];

								b = ucx * dx + ucy * dy + ucz * dz;
								b2 = ux * dx + uy * dy + uz * dz;

								d1 = sqrtf(tmp/*, 0.5f*/);
								dd5 = __fdividef(1.0f, tmp * tmp * d1);
								bb2d7 = 15.0f * b * b2 * __fdividef(dd5, tmp);
								umd5 = - 3.0f * (ux*ucx + uy*ucy + uz*ucz) * dd5;

								hx += (b * 3.0f * dx - tmp * ucx) * dd5;
								hy += (b * 3.0f * dy - tmp * ucy) * dd5;
								hz += (b * 3.0f * dz - tmp * ucz) * dd5;


								fx += -dx * (umd5 + bb2d7)
								+ 3.0f * (b * ux + b2 * ucx) * dd5;

								fy += -dy * (umd5  +  bb2d7)
								+ 3.0f * (b * uy + b2 * ucy) * dd5;

								fz += -dz * (umd5  +  bb2d7)
								+ 3.0f * (b * uz + b2 * ucz) * dd5;

								/*if (fx != fx || fy != fy || fz != fz) {	//nan
									printf("NAN in particle iteraction (Before LJ)[%d] and meta:[%d]\n", i, n);
									printf("x = %f, y = %f, z = %f,\n", px, py, pz);
									printf("x = %f, y = %f, z = %f,\n", xd[n], xd[n], xd[n]);
									printf("fx = %f, fy = %f, fz = %f,\n", fx, fy, fz);
									printf("hx = %f, hy = %f, hz = %f,\n", hx, hy, hz);
									if (fx != fx) fx = 0.0f;
									if (fy != fy) fy = 0.0f;
									if (fz != fz) fz = 0.0f;
								}*/


								/* Lennard Jonnes force is equal:
								 * flj = 24.0 * E * (2 * __powf(SIGMA2/D2, 6) - __powf(SIGMA2/D2, 3)) / d1;
								 * where SIGMA2 is SIGMA*SIGMA;
								 * But each component is equal to (for x) fx = dx  * flj / D1;
								 * the last D1 in were included to force flj to decrease calculations;
								 * */

								if (d1 < devCParBH.ljCutOffR * devCParBH.sigma) {
									pow3s2d2 = __powf(SQ(devCParBH.sigma)/tmp, 3);
									flj = 24.0f * devCParBH.eps * (pow3s2d2  * (2.0f * pow3s2d2 - 1.0f)) / tmp;
									flj -= devCParBH.ljCutOffForce;
									fx +=  dx * flj;
									fy +=  dy * flj;
									fz +=  dz * flj;

									/*if (fx != fx || fy != fy || fz != fz) {	//nan
										printf("NAN in particle iteraction (After LJ)[%d] and meta:[%d]. Flj = %f\n", i, n, flj);
										printf("x = %f, y = %f, z = %f,\n", px, py, pz);
										printf("x = %f, y = %f, z = %f,\n", xd[n], xd[n], xd[n]);
										printf("fx = %f, fy = %f, fz = %f,\n", fx, fy, fz);
										printf("hx = %f, hy = %f, hz = %f,\n", hx, hy, hz);
										if (fx != fx) fx = 0.0f;
										if (fy != fy) fy = 0.0f;
										if (fz != fz) fz = 0.0f;
									}*/
								}
							}
						} else {
							// push cell onto stack
							depth++;
							if (sbase == threadIdx.x) {
								node[depth] = n;
								pos[depth] = 0;
							}
						}
					} else {
						depth = max(j, depth - 1);	// early out because all remaining children are also zero
					}
				}
				depth--;	// done with this level
			}

		//	fz += 0.7f;

			if (devCParBH.cf == ConstParams::BARREL) {
				//making center (0;0)
				px = px - devCParBH.barrelR;
				py = py - devCParBH.barrelR;
				float xr, yr;
				//calculating nearest point of the circle to current particle
				if (fabs(px) < 1E-6) {
					xr = 0;
					if (py > 0) yr = devCParBH.barrelR;
					else yr = -devCParBH.barrelR;
				} else {
					xr = sqrtf(SQ(devCParBH.barrelR) / (1.0f + SQ(py/px))/*, 0.5f*/);

					//there are 2 roots, we should define + or -
					if (px < 0.0f) xr = -xr;

					yr = xr * py / px;
				}

				dx = px - xr;
				dy = py - yr;
				tmp = (dx * dx  + dy * dy );

#define walls 15.0f

				pow3s2d2 = __powf(SQ(devCParBH.sigmaWall) / tmp, 3);
				flj = 24.0f * devCParBH.epsWall * (pow3s2d2  * (2.0f * pow3s2d2 - 1.0f)) / tmp;
				fx +=  dx * flj;	//maybe should be -dx...
				fy +=  dy * flj;
				fz += 24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/pz, 12) - __powf(devCParBH.sigmaWall/pz, 6)) / pz;
				fz += -24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/(devCParBH.lz - pz), 12) - __powf(devCParBH.sigmaWall/(devCParBH.lz - pz), 6)) / (devCParBH.lz - pz);


				/*flj = walls / (expf(walls * sgn(-sqrtf(px * px + py * py) + devCParBH.barrelR) * sqrtf(fabs(tmp))) + 1.0f);
				fx +=  -(dx) * flj;
				fy +=  -(dy) * flj;

				flj = walls / (expf(walls * sgn(pz) * sqrtf(fabs(pz))) + 1.0f);
				if (flj != flj) printf("1FLJ is NAN: pz = %f\n", pz);
				fz += flj;
				flj = - walls / (expf(walls * sgn(devCParBH.lz - pz) * sqrtf(fabs(devCParBH.lz - pz))) + 1.0f);
				if (flj != flj) printf("2FLJ is NAN: pz = %f\n", pz);
				fz += flj;*/
			} else {

			/*	if (fx < 0) fx = fx / (expf(-walls * px) + 1.0f);
				else fx = fx / (expf(-walls * (devCParBH.lx - px)) + 1.0f);

				if (fy < 0) fy = fy / (expf(-walls * py) + 1.0f);
				else fy = fy / (expf(-walls * (devCParBH.ly - py)) + 1.0f);

				if (fz < 0) fz = fz / (expf(-walls * pz) + 1.0f);
				else fz = fz / (expf(-walls * (devCParBH.lz - pz)) + 1.0f);*/

				fx += 24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/px, 12) - __powf(devCParBH.sigmaWall/px, 6)) / px;
				fy += 24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/py, 12) - __powf(devCParBH.sigmaWall/py, 6)) / py;
				fz += 24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/pz, 12) - __powf(devCParBH.sigmaWall/pz, 6)) / pz;

				fx += -24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/(devCParBH.lx - px), 12) - __powf(devCParBH.sigmaWall/(devCParBH.lx - px), 6)) / (devCParBH.lx - px);
				fy += -24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/(devCParBH.ly - py), 12) - __powf(devCParBH.sigmaWall/(devCParBH.ly - py), 6)) / (devCParBH.ly - py);
				fz += -24.0f * devCParBH.epsWall * (2.0f * __powf(devCParBH.sigmaWall/(devCParBH.lz - pz), 12) - __powf(devCParBH.sigmaWall/(devCParBH.lz - pz), 6)) / (devCParBH.lz - pz);



		/*		flj = walls / (expf(walls * px) + 1.0f);
				if (flj != flj) printf("1FLJ is NAN: px = %f\n", px);
				fx += flj;
				flj = - walls / (expf(walls * (devCParBH.lx - px)) + 1.0f);
				if (flj != flj) printf("2FLJ is NAN: px = %f\n", px);
				fx += flj;

				flj = walls / (expf(walls * py) + 1.0f);
				if (flj != flj) printf("1FLJ is NAN: py = %f\n", py);
				fy += flj;
				flj = - walls / (expf(walls * (devCParBH.ly - py)) + 1.0f);
				if (flj != flj) printf("2FLJ is NAN: py = %f\n", py);
				fy += flj;

				flj = walls / (expf(walls * pz) + 1.0f);
				if (flj != flj) printf("1FLJ is NAN: pz = %f\n", pz);
				fz += flj;
				flj = - walls / (expf(walls * (devCParBH.lz - pz)) + 1.0f);
				if (flj != flj) printf("2FLJ is NAN: pz = %f\n", pz);
				fz += flj;*/

			}

			hx += devChParBH.hExtX;
			hy += devChParBH.hExtY;
			hz += devChParBH.hExtZ;

#define	NX (uy * hz - uz * hy)
#define	NY (uz * hx - ux * hz)
#define	NZ (ux * hy - uy * hx)

			deltaPhyd[i] = phy1d[i] * devChParBH.dTimeCurrent;
			deltaThetad[i] = theta1d[i] * devChParBH.dTimeCurrent;

			deltaXd[i] = x1d[i] * devChParBH.dTimeCurrent;
			deltaYd[i] = y1d[i] * devChParBH.dTimeCurrent;
			deltaZd[i] = z1d[i] * devChParBH.dTimeCurrent;
#define MAX_DELTA 0.1f
			if (fabs(deltaXd[i]) > MAX_DELTA) {
				printf("Too big dx[%d] = %f\n", i, deltaXd[i]);
//				deltaXd[i] = MAX_DELTA * deltaXd[i] /  fabs(deltaXd[i]);
				deltaXd[i] = 1E10;
			}
			if (fabs(deltaYd[i]) > MAX_DELTA) {
				printf("Too big dy[%d] = %f\n", i, deltaYd[i]);
//				deltaYd[i] = MAX_DELTA * deltaYd[i] /  fabs(deltaYd[i]);
				deltaYd[i] = 1E10;
			}
			if (fabs(deltaZd[i]) > MAX_DELTA) {
				printf("Too big dz[%d] = %f\n", i, deltaZd[i]);
//				deltaZd[i] = MAX_DELTA * deltaZd[i] /  fabs(deltaZd[i]);
				deltaZd[i] = 1E10;
			}

/*			if (devCParBH.gravitation == true) {
				fz += - devCParBH.r * g * (devCParBH.roParticles - devCParBH.roEnvironment)
						/ (4.0f * M_PI * devCParBH.myu * devCParBH.myu * myu0 / 3.0f);
			}*/

			if (fx != fx || fy != fy || fz != fz ||
				hx != hx || hy != hy || hz != hz) {	//nan
				printf("Force Kernel: NAN in particle[%d]\n", i);
				printf("x = %f, y = %f, z = %f,\n", px, py, pz);
				printf("fx = %f, fy = %f, fz = %f,\n", fx, fy, fz);
				printf("hx = %f, hy = %f, hz = %f,\n", hx, hy, hz);
/*				if (fx != fx) fx = 0.0f;
				if (fy != fy) fy = 0.0f;
				if (fz != fz) fz = 0.0f;
				if (hx != hx) hx = 0.0f;
				if (hy != hy) hy = 0.0f;
				if (hz != hz) hz = 0.0f;*/
				fx = 1E10;
				fy = 1E10;
				fz = 1E10;
				hx = 0.0f;
				hy = 0.0f;
				hz = 0.0f;
			}

			if (devCParBH.thermalBath == true) {
				localState = rndStatesd[i];

				phy1d[i] += 2.5f * ((NZ - phy1d[i] * devCParBH.nyu) * devChParBH.dTimeCurrent
						+ hiprand_normal(&localState) /* 3.0f * devCParBH.nyu*/ * devChParBH.sqrtdTime * devCParBH.qr);

				theta1d[i] += 2.5f * ((- NX * __sinf(phyd[i]) + NY * __cosf(phyd[i])
					- theta1d[i] * devCParBH.nyu) * devChParBH.dTimeCurrent
					+ hiprand_normal(&localState) /* 3.0f * devCParBH.nyu*/ * devChParBH.sqrtdTime * devCParBH.qr);

				x1d[i] += (fx - x1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent
						+ hiprand_normal(&localState) /* 2.0f * devCParBH.eta*/ * devChParBH.sqrtdTime * devCParBH.qt;
				y1d[i] += (fy - y1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent
					+ hiprand_normal(&localState) /* 2.0f * devCParBH.eta*/ * devChParBH.sqrtdTime * devCParBH.qt;
				z1d[i] += (fz - z1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent
					+ hiprand_normal(&localState) /* 2.0f * devCParBH.eta*/ * devChParBH.sqrtdTime * devCParBH.qt;

				rndStatesd[i] = localState;


				//if(i % 100 == 0) printf("x1[%d] = %f, y1[%d] = %f, z1[%d] = %f\n", i, x1d[i], i, y1d[i], i, z1d[i]);
				//printf("%d rnd %f\n", i, hiprand_normal(&localState));
			} else {
				phy1d[i] += 2.5f * (NZ - phy1d[i] * devCParBH.nyu) * devChParBH.dTimeCurrent;
				theta1d[i] += 2.5f * (- NX * __sinf(phyd[i]) + NY * __cosf(phyd[i])
					- theta1d[i] * devCParBH.nyu) * devChParBH.dTimeCurrent;

				x1d[i] += (fx - x1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent;
				y1d[i] += (fy - y1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent;
				z1d[i] += (fz - z1d[i] * devCParBH.eta) * devChParBH.dTimeCurrent;
				//if(i % 100 == 0) printf("x1[%d] = %f, y1[%d] = %f, z1[%d] = %f\n", i, x1d[i], i, y1d[i], i, z1d[i]);

			}
		}
	}
	//rndStatesd[IND] = localState;
}


/******************************************************************************/
/*** advance bodies ***********************************************************/
/******************************************************************************/

__device__
bool checkLocation(float x, float y, float z){
	if (devCParBH.cf == ConstParams::BARREL) {
		if (SQ(x - devCParBH.barrelR) + SQ(y - devCParBH.barrelR) < SQ(devCParBH.barrelR)
				&& z > 0.0f && z < devCParBH.lz) return true;
		else return false;
	} else {
		if (x > 0.0f && x < devCParBH.lx &&
			y > 0.0f && y < devCParBH.ly &&
			z > 0.0f && z < devCParBH.lz) return true;
		else return false;
	}
}

__device__
bool checkLocationFull(float x, float y, float z, int n){
	float xi, yi, zi;
	if (checkLocation(x, y, z) == false) return false;

	for (int i = 0; i < nbodiesd; i++){
		xi = xd[i];
		yi = yd[i];
		zi = zd[i];

		if (fabs(x - xi) < 2.0f &&
			fabs(y - yi) < 2.0f &&
			fabs(z - zi) < 2.0f &&
			i != n) {
			if (SQ(x - xi) + SQ(y - yi) + SQ(z - zi) < 4.0f) return false;	//4 = 2R * 2R = 4R = 4
		}
	}
	return true;
}

__global__
__launch_bounds__(THREADS6, FACTOR6)
void integrationKernel() {
	register int i, inc;
	register float deltaPhy, deltaTheta, deltaX, deltaY, deltaZ;
	register float x, y, z;
	// iterate over all bodies assigned to thread
	inc = blockDim.x * gridDim.x;
	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
		// integrate
		deltaPhy = deltaPhyd[i];
		deltaTheta = deltaThetad[i];
		deltaX = deltaXd[i];
		deltaY = deltaYd[i];
		deltaZ = deltaZd[i];

		if(deltaPhy != deltaPhy || deltaTheta != deltaTheta ||
				deltaX != deltaX || deltaY != deltaY || deltaZ != deltaZ) {
			printf("Integration kernel: NAN in particle[%d]\n", i);
			printf("x = %f, y = %f, z = %f,\n", xd[i], yd[i], zd[i]);
			printf("deltaX = %f, deltaY = %f, deltaZ = %f,\n", deltaX, deltaY, deltaZ);
			printf("deltaPhy = %f, deltaTheta = %f\n", deltaPhy, deltaTheta);
			if (deltaPhy != deltaPhy) deltaPhy = 0;
			if (deltaTheta != deltaTheta) deltaTheta = 0;
			if (deltaX != deltaX) deltaX = 1E10;	//move particle fare from current place
			if (deltaY != deltaY) deltaY = 1E10; //then it will be automatically located in a random place
			if (deltaZ != deltaZ) deltaZ = 1E10;
		}

		phyd[i] += deltaPhy;
		thetad[i] += deltaTheta;
		/*xd[i] += deltaX;
		yd[i] += deltaY;
		zd[i] += deltaZ;*/

		x = xd[i];
		y = yd[i];
		z = zd[i];

		x += deltaX;
		y += deltaY;
		z += deltaZ;

		if (checkLocation(x, y, z) == false){
			hiprandState localState;
			localState = rndStatesd[IND];

			do {
				x = 1.1f + hiprand_uniform(&localState) * (devCParBH.lx - 2.2f);
				y = 1.1f + hiprand_uniform(&localState) * (devCParBH.ly - 2.2f);
				z = 1.1f + hiprand_uniform(&localState) * (devCParBH.lz - 2.2f);
			} while(checkLocationFull(x, y, z, i) == false);

			printf("Particle[%d] out of borders: moved.\n", i);
			x1d[i] = 0.0f;
			y1d[i] = 0.0f;
			z1d[i] = 0.0f;
			rndStatesd[IND] = localState;
		}


		xd[i] = x;
		yd[i] = y;
		zd[i] = z;
	}
}

__global__
__launch_bounds__(THREADS0, FACTOR0)
void getCurrentPhysKernel(float* ringPhy) {
	register int i, inc;
	float x, y, phy;
	// iterate over all nodes assigned to thread
	inc = blockDim.x * gridDim.x;
	for (i = threadIdx.x + blockIdx.x * blockDim.x; i < nbodiesd; i += inc) {
		x = xd[i] - devCParBH.barrelR;
		y = yd[i] - devCParBH.barrelR;
		phy = atan2f(y, x);
		ringPhy[i] = phy;
	}
}

__global__
__launch_bounds__(THREADS0, FACTOR0)
void getRingDistrKernel(float* oldPhy, float* newPhy, int nRings, float* dphy, int* partPos) {
	float dPhyCached, r;
	int interval;
	int inc = blockDim.x * gridDim.x;
	//int ind = IND;
	float x, y;
	__shared__ float dPhyAddayShared[SHARED_ARRAY_BH];	//nRings should not be bigger than SHARED_ARRAY;
	__shared__ int sharedCounter[SHARED_ARRAY_BH];

	if (threadIdx.x < nRings) {
		dPhyAddayShared[threadIdx.x] = 0.0f;
		sharedCounter[threadIdx.x] = 0;
	}
	if (IND < nRings) {
		dphy[IND] = 0.0f;
		partPos[IND] = 0;
	}
	__syncthreads();

	for (int i = IND; i < nbodiesd; i += inc) {
		x = xd[i];
		y = yd[i];

		dPhyCached = newPhy[i] - oldPhy[i];
		if (dPhyCached > M_PI) dPhyCached -= 2.0f * M_PI;
		if (dPhyCached < -M_PI) dPhyCached += 2.0f * M_PI;

		r = sqrtf(SQ(x - devCParBH.barrelR) + SQ(y - devCParBH.barrelR)/*, 0.5f*/);
		interval = (int) floorf((float)nRings * r / devCParBH.barrelR);	//ly - is R of barrel
		atomicAdd(&(dPhyAddayShared[interval]), dPhyCached);
		atomicAdd(&(sharedCounter[interval]), 1);
	}
	__syncthreads();

	if (threadIdx.x == 0) {
		for (int i = 0; i < nRings; i++) {
			if (sharedCounter[i] > 0) atomicAdd(&(dphy[i]), dPhyAddayShared[i] / (sharedCounter[i] * gridDim.x));
			atomicAdd(&(partPos[i]), sharedCounter[i]);
		}
	}
}



static void CudaTest(char *msg) {
	hipError_t e;

	hipDeviceSynchronize();
	if (hipSuccess != (e = hipGetLastError())) {
		fprintf(stderr, "%s: %d\n", msg, e);
		fprintf(stderr, "%s\n", hipGetErrorString(e));
		exit(-1);
	}
}

void init(int blocks, hiprandState* devRndStates) {
	initializationKernel<<<blocks * FACTOR5, THREADS5>>>(devRndStates, time(NULL));
	CudaTest("init kernel launch failed");

	hipFuncSetCacheConfig(reinterpret_cast<const void*>(averageU), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(calcUKernel), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(boundingBoxKernel), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(boundingBoxKernel), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(treeBuildingKernel), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(summarizationKernel), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(sortKernel), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(forceCalculationKernel), hipFuncCachePreferL1);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(integrationKernel), hipFuncCachePreferL1);

	hipGetLastError();	// reset error value
}

void calcU(int blocks) {
	calcUKernel<<<blocks * FACTOR0, THREADS0>>>();
	CudaTest("kernel 0 launch failed");
	hipDeviceSynchronize();
}

void buildBox(int blocks) {
	boundingBoxKernel<<<blocks * FACTOR1, THREADS1>>>();
	CudaTest("kernel 1 launch failed");
	hipDeviceSynchronize();
}

void buildTree(int blocks) {
	treeBuildingKernel<<<blocks * FACTOR2, THREADS2>>>();
	CudaTest("kernel 2 launch failed");
	hipDeviceSynchronize();
}

void summarize(int blocks) {
	summarizationKernel<<<blocks * FACTOR3, THREADS3>>>();
	CudaTest("kernel 3 launch failed");
	hipDeviceSynchronize();
}

void sort(int blocks) {
	sortKernel<<<blocks * FACTOR4, THREADS4>>>();
	CudaTest("kernel 4 launch failed");
	hipDeviceSynchronize();
}

void force(int blocks) {
	forceCalculationKernel<<<blocks * FACTOR5, THREADS5>>>();
	CudaTest("kernel 5 launch failed");
	hipDeviceSynchronize();
}

void integrate(int blocks) {
	integrationKernel<<<blocks * FACTOR6, THREADS6>>>();
	CudaTest("kernel 6 launch failed");
	hipDeviceSynchronize();
}

void fillConstantPointers(int nbodies, int nnodes, float* mass, BHArrays arrl, Box boxl, PartParams devMatrixes) {
	float epssq = 0.05f * 0.05f;
	float itolsq = 1.0f / (0.5f * 0.5f);

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nnodesd), &nnodes, sizeof(int), 0, hipMemcpyHostToDevice))
		throw DeviceMemCpyToSymbolException("copying of nnodes to device failed\n");	//CudaTest("nnode copy to device failed");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(nbodiesd), &nbodies, sizeof(int), 0, hipMemcpyHostToDevice))
		throw DeviceMemCpyToSymbolException("copying of nnodes to device failed\n");	//CudaTest("nnode copy to device failed");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(epssqd), &epssq, sizeof(float), 0, hipMemcpyHostToDevice))
		throw DeviceMemCpyToSymbolException("copying of epssq to device failed\n");	//CudaTest("epssq copy to device failed");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(itolsqd), &itolsq, sizeof(float), 0, hipMemcpyHostToDevice))
		throw DeviceMemCpyToSymbolException("copying of itolsq to device failed\n");	//CudaTest("itolsq copy to device failed");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(errd), &(arrl.err), sizeof(void*), 0, hipMemcpyHostToDevice))
				throw DeviceMemCpyToSymbolException("copying of arrl.err to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(sortd), &(arrl.sort), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of arrl.err to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(childd), &(arrl.child), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of arrl.err to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(countd), &(arrl.count), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of arrl.err to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(startd), &(arrl.start), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of arrl.err to device failed\n");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(deltaPhyd), &(devMatrixes.deltaPhy), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaPhy to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(deltaThetad), &(devMatrixes.deltaTheta), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaTheta to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(phyd), &(devMatrixes.phy), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaPhy to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(thetad), &(devMatrixes.theta), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.Theta to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(phy1d), &(devMatrixes.phy1), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.Phy1 to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(theta1d), &(devMatrixes.theta1), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.Theta1 to device failed\n");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(deltaXd), &(devMatrixes.deltaX), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaX to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(deltaYd), &(devMatrixes.deltaY), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaY to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(deltaZd), &(devMatrixes.deltaZ), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaZ to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(xd), &(devMatrixes.x), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.x to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(yd), &(devMatrixes.y), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.y to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(zd), &(devMatrixes.z), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.z to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(x1d), &(devMatrixes.x1), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.x1 to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(y1d), &(devMatrixes.y1), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.y1 to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(z1d), &(devMatrixes.z1), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.z1 to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(uxd), &(devMatrixes.ux), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.ux to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(uyd), &(devMatrixes.uy), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.uy to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(uzd), &(devMatrixes.uz), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.uz to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(rndStatesd), &(devMatrixes.rndStates), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaPhy to device failed\n");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(massd), &(mass), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of devMatrixes.deltaPhy to device failed\n");

	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxxd), &(boxl.maxx), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxyd), &(boxl.maxy), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(maxzd), &(boxl.maxz), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minxd), &(boxl.minx), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minyd), &(boxl.miny), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
	if (hipSuccess != hipMemcpyToSymbol(HIP_SYMBOL(minzd), &(boxl.minz), sizeof(void*), 0, hipMemcpyHostToDevice))
			throw DeviceMemCpyToSymbolException("copying of boxl.maxx to device failed\n");
}


void fillGloabalChangableBH(ChangableParams* chPar) {
	hipMemcpyToSymbol(HIP_SYMBOL(devChParBH), chPar, sizeof(ChangableParams), 0, hipMemcpyHostToDevice);
}

void fillGloabalConstantBH(ConstParams* cPar) {
	hipMemcpyToSymbol(HIP_SYMBOL(devCParBH), cPar, sizeof(ConstParams), 0, hipMemcpyHostToDevice);
}

float3 averageU(int blocks) {
	float3 u;
	float3* devU;
	if (hipSuccess != hipMalloc((void**)&devU, sizeof(float3)))
		throw DeviceMemoryAllocationException ("devU Allocation exception");
	averageUKernel<<<blocks * FACTOR0, THREADS0>>>(devU);
	CudaTest("AverageU launch failed");
	if (hipSuccess != hipMemcpy(&u, devU, sizeof(float3), hipMemcpyDeviceToHost))
		throw DeviceMemoryCopyException ("devU copy exception");

	hipFree(devU);
	return u;
}

void getCurrentPhysBH(int blocks, float* devOldPhy) {
	getCurrentPhysKernel<<<blocks * FACTOR0, THREADS0>>> (devOldPhy);
}

void getRingStatBH(int blocks, int nRings, int nPart, float** devOldPhy, float* dphy, int* pd) {
	float* devNewPhy;
	float* devDPhy;
	int* devPD;
	float* tmp;

	if (hipSuccess != hipMalloc((void**)&devNewPhy, sizeof(float) * nPart))
		throw DeviceMemoryAllocationException("Error allocation of devNewPhy");
	if (hipSuccess != hipMalloc((void**)&devDPhy, sizeof(float) * nRings))
		throw DeviceMemoryAllocationException("Error allocation of devDPhy");
	if (hipSuccess != hipMalloc((void**)&devPD, sizeof(int) * nRings))
		throw DeviceMemoryAllocationException("Error allocation of devPD");

	getCurrentPhysBH(blocks, devNewPhy);
	getRingDistrKernel<<<blocks * FACTOR0, THREADS0>>>(*devOldPhy, devNewPhy, nRings, devDPhy, devPD);

	hipMemcpy(dphy, devDPhy, sizeof(float) * nRings, hipMemcpyDeviceToHost);
	hipMemcpy(pd, devPD, sizeof(int) * nRings, hipMemcpyDeviceToHost);

	tmp = *devOldPhy;
	*devOldPhy = devNewPhy;

	if (hipSuccess != hipFree(tmp))
		throw DeviceMemoryException(std::string("Error free of tmp") +
				std::string(hipGetErrorString(hipGetLastError())));
	if (hipSuccess != hipFree(devDPhy))
		throw DeviceMemoryException(std::string("Error free of devDPhy") +
				std::string(hipGetErrorString(hipGetLastError())));
	if (hipSuccess != hipFree(devPD))
		throw DeviceMemoryException(std::string("Error free of devPD") +
				std::string(hipGetErrorString(hipGetLastError())));
}
